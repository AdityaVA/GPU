/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use cudaFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    //A[N*i+j] = A[N*i+j]*B[N*j+i]
    int id = (blockIdx.y * gridDim.x + blockIdx.x)*blockDim.x*blockDim.y 
    + threadIdx.y*blockDim.x + threadIdx.x;
    int i = id/N;
    int j = id%N;
    if(id<N*N)
    {
        A[N*i+j] = A[N*i+j]*B[N*j+i];
    }
}

__global__
void FindWeightMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    int id = (blockIdx.y * gridDim.x + blockIdx.x)*blockDim.x*blockDim.y 
    + threadIdx.y*blockDim.x + threadIdx.x;
    if(id<N*N)
    {
        A[id] = max(A[id],B[id]);
    }
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    int id = (blockIdx.y * gridDim.x + blockIdx.x)*blockDim.x*blockDim.y 
    + threadIdx.y*blockDim.x + threadIdx.x;
    
    if(id < 2 * N * 2 * N)
    {
        int i = id/(2*N);
        int j = id%(2*N);
        int weight_i = i%N;
        int weight_j = j%N;
        // printf("%d %d %d %d %d %d\n",id,i,j,weight_i,weight_j,N);
        B[2*N*i+j] = A[N*weight_i+weight_j]*B[2*N*i+j];
    }
}


int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);
    //first copy A, B matrices to device memory
    hipMalloc(&d_A, N * N * sizeof(long int));
    hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_B, N * N * sizeof(long int));
    hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipFree(d_B);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;


    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);
    //Now copy C as well to perform some operations
    hipMalloc(&d_C, N * N * sizeof(long int));
    hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);

    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipFree(d_C);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;


    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    hipMalloc(&d_D, 4 * N * N * sizeof(long int));
    hipMemcpy(d_D, D, 4 * N * N * sizeof(long int), hipMemcpyHostToDevice);
    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;
    hipFree(d_A);
    
    
    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}