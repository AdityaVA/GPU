/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__
void convolve(long int* d_mat, long int* d_ans, long int* d_filter, int m, int n, int k) {

    
    int id = (blockIdx.y * gridDim.x + blockIdx.x)*blockDim.x*blockDim.y 
    + threadIdx.y*blockDim.x + threadIdx.x;
    extern __shared__ long int filter[];
    if(threadIdx.x == 0)
    {
        for(int i = 0; i < k; i++)
        {
            for(int j = 0; j < k; j++)
            {
                filter[i*k+j] = d_filter[i*k + j];
            }
        }
        // printf("Input : %d %d %d\n", m, n, k);
    }

    __syncthreads();

    int i = id / n;
    int j = id % n;

    if(i < m && j < n)
    {
        long int sum = 0;
        for(int x = 0; x < k; x++)
        {
            for(int y = 0; y < k; y++)
            {
                int cur_i = i - k/2 + x;
                int cur_j = j - k/2 + y;
                if(cur_i >= 0 && cur_i < m && cur_j >= 0 && cur_j < n)
                {
                    sum += d_mat[cur_i * n + cur_j] * filter[x * k + y];
                }
            }
        }
        d_ans[i * n + j] = sum;
    }

}

int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    // dim3 threadsPerBlock = dim3(32, 32, 1);
    // dim3 blocksPerGrid = dim3(ceil(n * m / 1024.0), 1, 1);
    dim3 threadsPerBlock = dim3(n, 1, 1);
    dim3 blocksPerGrid = dim3(m, 1, 1);
    long int* d_mat;
    long int* d_filter;
    long int* d_ans;

    hipMalloc(&d_mat, m * n * sizeof(long int));
    hipMalloc(&d_filter, k * k * sizeof(long int));
    hipMalloc(&d_ans, m * n * sizeof(long int));

    hipMemcpy(d_mat, h_mat, m * n * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, k * k * sizeof(long int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch

    convolve<<<blocksPerGrid, threadsPerBlock, k * k * sizeof(long int)>>>(d_mat, d_ans, d_filter, m, n, k);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    
    hipMemcpy(h_ans, d_ans, m * n * sizeof(long int), hipMemcpyDeviceToHost);
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}