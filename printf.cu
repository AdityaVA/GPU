#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel() {
    printf("Hello World.\n");
}
int main() {
    dkernel<<<1, 1>>>();
    printf("CPU one\n");
    dkernel<<<1, 1>>>();
    printf("CPU two\n");
    dkernel<<<1, 1>>>();
    printf("CPU three\n");
	hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    printf("ON CPU\n");
    return 0;
}